/************************************************

CS6310 Artificial Neural Networks
Assignment 1

Author: Arulkumar S (CS15S023)

*************************************************/

#include<stdio.h>
#include "hip/hip_runtime.h"


#include <stdarg.h>     /* va_list, va_start, va_arg, va_end */
#include <stdlib.h>     /* srand, rand */
#include <time.h>       /* time */

FILE* fp;

/**
*  The instructions which will be running in NVIDIA graphics card processor cores.
*
*/
__global__ void multiply(double* matrixA, double* matrixB, int matrixMultInvariant, double* matrixResult)
{
	int matrix1RowStartIndex = blockIdx.x * matrixMultInvariant;
	int matrix2ColumnStartIndex = threadIdx.x * matrixMultInvariant;  // matrixMultInvariant is same as matrix1Column, matrix2Row
	int resultIndex = (blockIdx.x * blockDim.x) + threadIdx.x;
	double multiplicationValue = 0;

	for(int matrix1Index = matrix1RowStartIndex, matrix2Index = matrix2ColumnStartIndex; matrix1Index < (matrix1RowStartIndex + matrixMultInvariant) && matrix2Index < (matrix2ColumnStartIndex + matrixMultInvariant); matrix1Index++, matrix2Index++)
	{
		multiplicationValue += (matrixA[matrix1Index] * matrixB[matrix2Index]);
	}

	matrixResult[resultIndex] = multiplicationValue;

}

/*
 A wrapper to allocate memory in GPU device

 parameters:

 deviceMemoryAddress[in] = contains destination address in Device
 dataLength[in]			 = length of the array to be allocated

*/
template<typename T>
void allocateDeviceMemory(T** deviceMemoryAddress, int dataLength)
{
	int totalSize = dataLength * sizeof(T);
	hipMalloc((void**) deviceMemoryAddress, totalSize);
}


/*
 A wrapper to copy the memory contents from Host to GPU device

 parameters:

 hostMemory[in]			 = contains source address in Host
 deviceMemoryAddress[in] = contains destination address in Device
 dataLength[in]			 = length of the array to be copied

*/
template<typename T>
void copyToDevice(T* hostMemory, T** deviceMemoryAddress, int dataLength)
{
	//allocate the memory in device
	allocateDeviceMemory<T>(deviceMemoryAddress, dataLength);

	// copy the contents from Host to device
	hipMemcpy(*deviceMemoryAddress, hostMemory, dataLength * sizeof(T), hipMemcpyHostToDevice);
}

/*
 A wrapper to copy the memory contents from GPU Device to Host

 parameters:

 hostMemory[in]			 = contains Destination address in Host
 deviceMemoryAddress[in] = contains Source address in Device
 dataLength[in]			 = length of the array to be copied

*/

template<typename T>
void copyFromDevice(T* hostMemory, T* deviceMemoryAddress, int dataLength)
{
	int totalSize = dataLength * sizeof(T);
	
	// copy the results from device to host
	hipMemcpy(hostMemory, deviceMemoryAddress, totalSize, hipMemcpyDeviceToHost);;
}

/*
 A wrapper to free the memory allocate in GPU device.
 This API accepts variable number of arguments. Number of arguments should be given as first parameter.

 parameters:

 n_args[in] = number of arguments passed

*/
void freeDeviceMemory(int n_args, ...)
{
	// initialize variable argument list
	va_list vl;
	va_start(vl, n_args);
	void* pointer;

	for(int index= 0; index < n_args; index++)
	{
		pointer = va_arg(vl, void*);
		hipFree(pointer);
	}

	//end the variable argument list
	va_end(vl);
}

/*
fill the matrix in row major form randomly
*/
void fillMatrixRowMajor(double* matrix, int matrixRow, int matrixColumn)
{
	for(int rowIndex = 0; rowIndex < matrixRow; rowIndex++)
	{
		for(int columnIndex = 0; columnIndex < matrixColumn; columnIndex++)
		{
			matrix[(rowIndex * matrixColumn) + columnIndex] = rand() % 100;
		}
	}
}

/*
fill the matrix in column major form randomly
*/
void fillMatrixColumnMajor(double* matrix, int matrixRow, int matrixColumn)
{
	for(int rowIndex = 0; rowIndex < matrixRow; rowIndex++)
	{
		for(int columnIndex = 0; columnIndex < matrixColumn; columnIndex++)
		{
			matrix[(columnIndex * matrixRow) + rowIndex] = rand() % 10;
		}
	}
}


/*
 API to print the matrix
*/
void printMatrixRowMajor(char* name, double* matrix, int matrixRow, int matrixColumn)
{
	fprintf(fp, "\n%s (%d x %d)\n\n", name, matrixRow, matrixColumn);

	for(int rowIndex = 0; rowIndex < matrixRow; rowIndex++)
	{
		for(int columnIndex = 0; columnIndex < matrixColumn; columnIndex++)
		{
			fprintf(fp, "%-10.2f", matrix[(rowIndex * matrixColumn) + columnIndex]);
		}
		fprintf(fp, "\n");
	}

	fprintf(fp, "\n");
}

/*
 API to print the matrix
*/
void printMatrixColumnMajor(char* name, double* matrix, int matrixRow, int matrixColumn)
{
	fprintf(fp, "\n%s (%d x %d)\n\n", name, matrixRow, matrixColumn);

	for(int rowIndex = 0; rowIndex < matrixRow; rowIndex++)
	{
		for(int columnIndex = 0; columnIndex < matrixColumn; columnIndex++)
		{
			fprintf(fp, "%-10.2f", matrix[(columnIndex * matrixRow) + rowIndex]);
		}
		fprintf(fp, "\n");
	}

	fprintf(fp, "\n");
}

/*

Matrix multiplication in Host CPU

*/
void multiplyMatrixInHost(double* matrixA, double* matrixB, int matrix1Row, int matrixMultInvariant, int matrix2Column, double* matrixResult)
{

	// loop for all rows of matrixA
	for(int rowIndex = 0; rowIndex < matrix1Row; rowIndex++)
	{
		// loop for all columns of matrixA
		for(int colIndex = 0; colIndex < matrix2Column; colIndex++)
		{
			matrixResult[(rowIndex * matrix2Column) + colIndex] = 0;

			// loop for all internal elements (columns of matrixA, rows of matrixB)
			for(int internalIndex = 0; internalIndex < matrixMultInvariant; internalIndex++)
			{
				matrixResult[(rowIndex * matrix2Column) + colIndex] += (matrixA[(rowIndex * matrixMultInvariant) + internalIndex] * matrixB[(colIndex * matrixMultInvariant) + internalIndex]);
			}
		}
	}
}


/*
*  Matrix multiplication using NVIDIA Graphics card + CUDA C programming  
*
*  Main program entry
*/
int main()
{
	clock_t start, end;
	fp = fopen("Task2_data.txt", "w+");

	/* initialize random seed: */
	srand ((unsigned int)time(NULL));

	int matrix1Row, matrix1Column; 
	int matrix2Row, matrix2Column;  

	//get the total number of columns , rows count of matrices
	printf("\n row count of Matrix-1    : ");
	scanf("%d", &matrix1Row);
	printf("\n column count of Matrix-1 : ");
	scanf("%d", &matrix1Column);
	printf("\n row count of Matrix-2    : ");
	scanf("%d", &matrix2Row);
	printf("\n column count of Matrix-2 : ");
	scanf("%d", &matrix2Column);

	// if the matrices cannot be multiplied, through error and exit the program
	if(matrix1Column != matrix2Row)
	{
		printf("\n\nColumn count of Matrix-1 is not equal to Row count of Matrix-2. So, matrices cannot be multiplied!\n\n");
		fprintf(fp, "\n\nColumn count of Matrix-1 is not equal to Row count of Matrix-2. So, matrices cannot be multiplied!\n\n");

		printf("program will exit now!\n\n");

		getchar();

		fclose(fp);
		exit(0);
	}

	// assume that the matrix1 is stored in row-major format
	double* matrix1; matrix1 = (double*) malloc(matrix1Row * matrix1Column * sizeof(double));
	fillMatrixRowMajor(matrix1, matrix1Row, matrix1Column);

	// assume that the matrix2 is stored in column-major format
	double* matrix2; matrix2 = (double*) malloc(matrix2Row * matrix2Column * sizeof(double));
	fillMatrixColumnMajor(matrix2, matrix2Row, matrix2Column);

	// matrix multiplication result
	double* matrixMultiResult;
	matrixMultiResult = (double*) malloc(matrix1Row * matrix2Column * sizeof(double));

	printMatrixRowMajor("Matrix1", matrix1, matrix1Row, matrix1Column);
	printMatrixColumnMajor("Matrix2", matrix2, matrix2Row, matrix2Column);

	double* matrix1_d;
	double* matrix2_d;
	double* matrixMultiResult_d;

	//copy the input contents to Device
	copyToDevice<double>(matrix1, &matrix1_d, matrix1Row * matrix1Column);
	copyToDevice<double>(matrix2, &matrix2_d, matrix2Row * matrix2Column);
	allocateDeviceMemory<double>(&matrixMultiResult_d, matrix1Row * matrix2Column);

	//MULTIPLICATION IN GPU
	start = clock();
	multiply<<<matrix1Row, matrix2Column>>>(matrix1_d, matrix2_d, matrix1Column, matrixMultiResult_d);
	end = clock() - start;
	// copy result from device to host
	copyFromDevice<double>(matrixMultiResult, matrixMultiResult_d, matrix1Row * matrix2Column);
	printMatrixRowMajor("Multiplication result from GPU Device", matrixMultiResult, matrix1Row, matrix2Column);
	fprintf(fp, "GPU time taken = %f\n\n", (double) end / (double) CLOCKS_PER_SEC);
	printf("GPU time taken = %f\n\n", (double) end / (double) CLOCKS_PER_SEC);

	//MULTIPLICATION IN HOST
	start = clock();
	//do multiplication in Host
	multiplyMatrixInHost(matrix1, matrix2, matrix1Row, matrix1Column, matrix2Column, matrixMultiResult);
	end = clock() - start;
	printMatrixRowMajor("Multiplication result from Host", matrixMultiResult, matrix1Row, matrix2Column);
	printf( "PC time taken = %f\n\n", (double) end / (double) CLOCKS_PER_SEC);
	fprintf(fp, "PC time taken = %f\n\n", (double) end / (double) CLOCKS_PER_SEC);

	//wait for user key
	free(matrixMultiResult); free(matrix1); free(matrix2);
	freeDeviceMemory(3, matrix1_d, matrix2_d, matrixMultiResult_d);

	fclose(fp);

	getchar(); getchar();
	return 0;
}