/************************************************

CS6310 Artificial Neural Networks
Assignment 1

Vector Dot product

Author: Arulkumar S (CS15S023)

*************************************************/

#include <stdio.h>
#include <hip/hip_runtime.h>


#include <stdarg.h>     /* va_list, va_start, va_arg, va_end */
#include <stdlib.h>     /* srand, rand */
#include <time.h>
#define NUMBER_OF_THREADS  512

FILE* fp;

/**
*  The instructions which will be running in NVIDIA graphics card processor cores.
*
*/
__global__ void multiply(int* number1, int* number2, int* products, int length)
{
	int index = (blockIdx.x * NUMBER_OF_THREADS) + threadIdx.x;

	// multiply the components of the vector
	if(index < length)
		products[index] = number1[index] * number2[index];

}

/*
 A wrapper to copy the memory contents from Host to GPU device

 parameters:

 hostMemory[in]			 = contains source address in Host
 deviceMemoryAddress[in] = contains destination address in Device
 dataLength[in]			 = length of the array to be copied

*/
template<typename T>
void copyToDevice(T* hostMemory, T** deviceMemoryAddress, int dataLength)
{
	int totalSize = dataLength * sizeof(T);
	hipMalloc((void**) deviceMemoryAddress, totalSize);
	hipMemcpy(*deviceMemoryAddress, hostMemory, totalSize, hipMemcpyHostToDevice);
}

/*
 A wrapper to copy the memory contents from GPU Device to Host

 parameters:

 hostMemory[in]			 = contains Destination address in Host
 deviceMemoryAddress[in] = contains Source address in Device
 dataLength[in]			 = length of the array to be copied

*/

template<typename T>
void copyFromDevice(T* hostMemory, T* deviceMemoryAddress, int dataLength)
{
	int totalSize = dataLength * sizeof(T);
	
	// copy the results from device to host
	hipMemcpy(hostMemory, deviceMemoryAddress, totalSize, hipMemcpyDeviceToHost);;
}

/*
 A wrapper to free the memory allocate in GPU device.
 This API accepts variable number of arguments. Number of arguments should be given as first parameter.

 parameters:

 n_args[in] = number of arguments passed

*/
void freeDeviceMemory(int n_args, ...)
{
	// initialize variable argument list
	va_list vl;
	va_start(vl, n_args);
	void* pointer;

	for(int index= 0; index < n_args; index++)
	{
		pointer = va_arg(vl, void*);
		hipFree(pointer);
	}

	//end the variable argument list
	va_end(vl);
}

/*
	randomly fill the vector to the given length
*/
void fillVector(char* name, int* vector, int length)
{
	fprintf(fp, "\n\n%s (%d x 1)\n\n", name, length);

	for(int index = 0; index < length; index++)
	{
		vector[index] = rand() % 10;
		fprintf(fp, "%d  ", vector[index]);
	}

	fprintf(fp, "\n\n");
}

/*
*  Vector dot product using NVIDIA Graphics card + CUDA C programming  
*
*  Main program entry
*/
int main()
{
	clock_t start, end;
	fp = fopen("Task1_data.txt", "w+");

	//based on the input format , change the vector preparation
	printf("Enter dimension (number of components) of vector space (e.g., 1000) : ");
	int veclength;
	scanf("%d", &veclength);

	int* vector1;
	vector1 = (int*) malloc(veclength * sizeof(int));
	fillVector("vector1", vector1, veclength);  //fill random numbers for vector1

	int* vector2;
	vector2 = (int*) malloc(veclength * sizeof(int));
	fillVector("vector2", vector2, veclength);  //fill random numbers for vector2

	int* products;
	products = (int*) malloc(veclength * sizeof(int));  //buffer to hold inner products

	int finalDotProduct = 0;

	int* vector1_d;
	int* vector2_d;
	int* products_d;

	// copy the input contents from Host to device
	copyToDevice<int>(vector1, &vector1_d, veclength);
	copyToDevice<int>(vector2, &vector2_d, veclength);
	copyToDevice<int>(products, &products_d, veclength);

	start = clock();
	// call function with 1 block, number of threads equal to total number of vector components
	multiply<<<(veclength / NUMBER_OF_THREADS) + 1, NUMBER_OF_THREADS, veclength * sizeof(int)>>>(vector1_d, vector2_d, products_d, veclength);
	end = clock() - start;

	fprintf(fp,"GPU time taken = %f\n\n", (double) end / (double) CLOCKS_PER_SEC);

	// copy the results from device to host
	copyFromDevice<int>(products, products_d, veclength);

	for(int index = 0; index < veclength; index++)
	{
		finalDotProduct += products[index];
	}


	// display the result
	printf("GPU dot product: %d\n\n", finalDotProduct);
	fprintf(fp, "GPU dot product: %d\n\n", finalDotProduct);

	//free device memory
	freeDeviceMemory(3, vector1_d, vector2_d, products_d);

	//do multiplication in PC
	finalDotProduct = 0;
	start = clock();
	for(int index = 0; index < veclength; index++)
	{
		finalDotProduct += (vector1[index] * vector2[index]);
	}

	end = clock() - start;
	fprintf(fp, "PC time taken = %f\n\n", (double) end / (double) CLOCKS_PER_SEC);

	// display the result
	printf("Host dot product: %d\n\n", finalDotProduct);
	fprintf(fp, "Host dot product: %d\n\n", finalDotProduct);

	//free host variables
	free(vector1); free(vector2); free(products);

	printf("\nrefer to Task1_data.txt for results\n\n"); 

	fclose(fp);

	//wait for user key
	getchar(); getchar();

	exit(0);
}