/************************************************

CS6310 Artificial Neural Networks
Assignment 1

Colvolution of 2D matrices

Author: Arulkumar S (CS15S023)

*************************************************/

#include<stdio.h>
#include "hip/hip_runtime.h"


#include <stdarg.h>     /* va_list, va_start, va_arg, va_end */
#include <stdlib.h>     /* srand, rand */
#include <time.h>       /* time */

FILE* fp;

/*
A wrapper to allocate memory in GPU device

parameters:

deviceMemoryAddress[in] = contains destination address in Device
dataLength[in]			 = length of the array to be allocated

*/
template<typename T>
void allocateDeviceMemory(T** deviceMemoryAddress, int dataLength)
{
	int totalSize = dataLength * sizeof(T);
	hipMalloc((void**) deviceMemoryAddress, totalSize);
}


/*
A wrapper to copy the memory contents from Host to GPU device

parameters:

hostMemory[in]			 = contains source address in Host
deviceMemoryAddress[in] = contains destination address in Device
dataLength[in]			 = length of the array to be copied

*/
template<typename T>
void copyToDevice(T* hostMemory, T** deviceMemoryAddress, int dataLength)
{
	//allocate the memory in device
	allocateDeviceMemory<T>(deviceMemoryAddress, dataLength);

	// copy the contents from Host to device
	hipMemcpy(*deviceMemoryAddress, hostMemory, dataLength * sizeof(T), hipMemcpyHostToDevice);
}

/*
A wrapper to copy the memory contents from GPU Device to Host

parameters:

hostMemory[in]			 = contains Destination address in Host
deviceMemoryAddress[in] = contains Source address in Device
dataLength[in]			 = length of the array to be copied

*/

template<typename T>
void copyFromDevice(T* hostMemory, T* deviceMemoryAddress, int dataLength)
{
	int totalSize = dataLength * sizeof(T);

	// copy the results from device to host
	hipMemcpy(hostMemory, deviceMemoryAddress, totalSize, hipMemcpyDeviceToHost);;
}

/*
A wrapper to free the memory allocate in GPU device.
This API accepts variable number of arguments. Number of arguments should be given as first parameter.

parameters:

n_args[in] = number of arguments passed

*/
void freeDeviceMemory(int n_args, ...)
{
	// initialize variable argument list
	va_list vl;
	va_start(vl, n_args);
	void* pointer;

	for(int index= 0; index < n_args; index++)
	{
		pointer = va_arg(vl, void*);
		hipFree(pointer);
	}

	//end the variable argument list
	va_end(vl);
}

/*
An API to check if the passed index is valid according to the total rows and columns

parameters:

currentRow[in] = row number for whcih the validness to be checked
currentColumn[in] = column number for whcih the validness to be checked
totalRow[in] = total number of rows
totalColumn[in] = total number of columns

*/
__host__ __device__ bool isIndexValid(int currentRow, int currentColumn, int totalRow, int totalColumn)
{
	bool isValid = false;

	//the index should be between 0 & (totalrows - 1) (or) (totalcolumns - 1) 
	if((currentRow >= 0)  && (currentRow < totalRow) && (currentColumn >= 0)  && (currentColumn < totalColumn))
		isValid = true;

	return isValid;
}

// a macro to simplify 2D array access by rows and columns
#define arrayAccess(arrayName, row, colCount, col) arrayName[(row * colCount) + col]

/*
the device API to convolute the big matrix with the given kernel

parameters:

inputMatrix[in]   = big matrix to be convoluted
kernel[in]        = the kernel which used to convolute big matrix
kernelRow[in]     = number of rows in kernel
kernelColumn[in]  = number of columns in kernel
inputColumn[in]   = number of columns in input matrix
the number of rows will be found by blockIdm.x
outputMatrix[out] = buffer to hold the convoluted matrix contents

*/
__global__ void convolute(int* inputMatrix, int* kernel, int kernelRow, int kernelColumn, int inputColumn, int* outputMatrix)
{
	//find the index of the input matrix to which the convolution has to be found
	int currentInputRowIndex = blockIdx.x;
	int currentInputColIndex = threadIdx.x;

	// find the midpoint of the kernel
	int rowMid = kernelRow / 2;
	int colMid = kernelColumn / 2;

	//for each value of the kernel, multiply it with appropriate input matrix value
	int value = 0;
	for (int index1 = -rowMid; index1 <= rowMid; index1++)
	{

		for (int index2 = -colMid; index2 <= colMid; index2++)
		{
			//find current kernel row & column index
			int currentKernelRow = (kernelRow - rowMid - 1) - index1;
			int currentKernelCol = (kernelColumn - colMid - 1) - index2;

			// find appropriate input matrix indexes w.r.t., kernel
			int inputRow = currentInputRowIndex + index1;
			int inputCol = currentInputColIndex + index2;

			// if the index of input matrix is valid, multiply it with kernel value
			if(isIndexValid(inputRow, inputCol, blockDim.x, inputColumn))
			{
				value += (arrayAccess(inputMatrix, inputRow, inputColumn, inputCol)
					* arrayAccess(kernel, currentKernelRow, kernelColumn, currentKernelCol));

			}

		}
	}

	// assign the new value in output matrix for particular index
	arrayAccess(outputMatrix, currentInputRowIndex, blockDim.x, currentInputColIndex) = value;
}

/*
the device API to convolute the big matrix with the given kernel (the multiplication is done as like filtering)

parameters:

inputMatrix[in]   = big matrix to be convoluted
kernel[in]        = the kernel which used to convolute big matrix
kernelRow[in]     = number of rows in kernel
kernelColumn[in]  = number of columns in kernel
inputColumn[in]   = number of columns in input matrix
the number of rows will be found by blockIdm.x
outputMatrix[out] = buffer to hold the convoluted matrix contents

*/
__global__ void convoluteAsFilter(int* inputMatrix, int* kernel, int kernelRow, int kernelColumn, int inputColumn, int* outputMatrix)
{
	//find the index of the input matrix to which the convolution has to be found
	int currentInputRowIndex = blockIdx.x;
	int currentInputColIndex = threadIdx.x;

	// find the midpoint of the kernel
	int rowMid = kernelRow / 2;
	int colMid = kernelColumn / 2;

	//for each value of the kernel, multiply it with appropriate input matrix value
	int value = 0;
	for (int index1 = -rowMid; index1 <= rowMid; index1++)
	{

		for (int index2 = -colMid; index2 <= colMid; index2++)
		{
			//find current kernel row & column index
			int currentKernelRow = rowMid + index1;
			int currentKernelCol = colMid + index2;

			// find appropriate input matrix indexes w.r.t., kernel
			int inputRow = currentInputRowIndex + index1;
			int inputCol = currentInputColIndex + index2;

			// if the index of input matrix is valid, multiply it with kernel value
			if(isIndexValid(inputRow, inputCol, blockDim.x, inputColumn))
			{
				value += (arrayAccess(inputMatrix, inputRow, inputColumn, inputCol)
					* arrayAccess(kernel, currentKernelRow, kernelColumn, currentKernelCol));

			}

		}
	}

	// assign the new value in output matrix for particular index
	arrayAccess(outputMatrix, currentInputRowIndex, blockDim.x, currentInputColIndex) = value;
}


/*
the host API to convolute the big matrix with the given kernel (the multiplication is done as like filtering)

parameters:

inputMatrix[in]   = big matrix to be convoluted
inputMatRow[in]   = the row count of the big matrix
inputMatCol[in]   = the column count of the big matrix
kernel[in]        = the kernel which used to convolute big matrix
kernelRow[in]     = number of rows in kernel
kernelColumn[in]  = number of columns in kernel
outputMatrix[out] = buffer to hold the convoluted matrix contents

*/
void convoluteInHost(int* inputMatrix, int inputMatRow, int inputMatColumn, int* kernel, int kernelRow, int kernelColumn, int* outputMatrix)
{

	for(int bigMatRow = 0; bigMatRow < inputMatRow; bigMatRow++)
	{
		for(int bigMatCol = 0; bigMatCol < inputMatColumn; bigMatCol++)
		{
			//find the index of the input matrix to which the convolution has to be found
			int currentInputRowIndex = bigMatRow;
			int currentInputColIndex = bigMatCol;

			// find the midpoint of the kernel
			int rowMid = kernelRow / 2;
			int colMid = kernelColumn / 2;

			//for each value of the kernel, multiply it with appropriate input matrix value
			int value = 0;
			for (int index1 = -rowMid; index1 <= rowMid; index1++)
			{

				for (int index2 = -colMid; index2 <= colMid; index2++)
				{
					//find current kernel row & column index
					int currentKernelRow = rowMid + index1;
					int currentKernelCol = colMid + index2;

					// find appropriate input matrix indexes w.r.t., kernel
					int inputRow = currentInputRowIndex + index1;
					int inputCol = currentInputColIndex + index2;

					// if the index of input matrix is valid, multiply it with kernel value
					if(isIndexValid(inputRow, inputCol, inputMatRow, inputMatColumn))
					{
						value += (arrayAccess(inputMatrix, inputRow, inputMatColumn, inputCol)
							* arrayAccess(kernel, currentKernelRow, kernelColumn, currentKernelCol));

					}

				}
			}

			// assign the new value in output matrix for particular index
			arrayAccess(outputMatrix, currentInputRowIndex, inputMatRow, currentInputColIndex) = value;
		}
	}
}



/*
API to print the matrix
*/
void printMatrixRowMajor(char* name, int* matrix, int matrixRow, int matrixColumn)
{
	fprintf(fp, "\n%s (%d x %d)\n\n", name, matrixRow, matrixColumn);

	for(int rowIndex = 0; rowIndex < matrixRow; rowIndex++)
	{
		for(int columnIndex = 0; columnIndex < matrixColumn; columnIndex++)
		{
			fprintf(fp, "%-10d", matrix[(rowIndex * matrixColumn) + columnIndex]);
		}
		fprintf(fp, "\n");
	}

	fprintf(fp, "\n");
}

/*
fill the matrix in row major form randomly
*/
void fillMatrixRowMajor(int* matrix, int matrixRow, int matrixColumn)
{
	for(int rowIndex = 0; rowIndex < matrixRow; rowIndex++)
	{
		for(int columnIndex = 0; columnIndex < matrixColumn; columnIndex++)
		{
			matrix[(rowIndex * matrixColumn) + columnIndex] = rand() % 10;
		}
	}
}


/*
Main entry point
*/

int main()
{
	clock_t start, end;
	fp = fopen("Task3_data.txt", "w+");

	/* initialize random seed: */
	srand ((unsigned int)time(NULL));

	//input variables
	int inputRow, inputColumn;
	int kernelRow, kernelColumn;

	//get the total number of columns , rows count of matrices
	printf("\n row count of input (Big) matrix    : ");
	scanf("%d", &inputRow);
	printf("\n column count of input (Big) matrix : ");
	scanf("%d", &inputColumn);
	printf("\n row count of Filter Matrix   : ");
	scanf("%d", &kernelRow);
	printf("\n column count of Filter Matrix : ");
	scanf("%d", &kernelColumn);

	//vlaidate for count of rows & columns for the filter
	if(((kernelRow % 2) != 1) ||  ((kernelColumn % 2) != 1))
	{
		fprintf(fp, "\n\nERROR: Kernel (Filter) rows (given: %d) and columns(given: %d) count should be an odd number.\n\n", kernelRow, kernelColumn);
		printf("\n\nERROR: Kernel (Filter) rows (given: %d) and columns(given: %d) count should be an odd number.\n\n", kernelRow, kernelColumn);

		getchar();getchar();
		exit(0);

	}

	// input data
	int* inputMatrix; inputMatrix = (int*)malloc(inputRow * inputColumn * sizeof(int));
	fillMatrixRowMajor(inputMatrix, inputRow, inputColumn);

	int* kernel; kernel = (int*)malloc(kernelRow * kernelColumn * sizeof(int));
	fillMatrixRowMajor(kernel, kernelRow, kernelColumn);

	// print the input and kernel matrix
	printMatrixRowMajor("input matrix", inputMatrix, inputRow, inputColumn);
	printMatrixRowMajor("kernel matrix", kernel, kernelRow, kernelColumn);

	int* outputMatrix;

	// declarations to hold kernel allocated memory
	int* inputMatrix_d;
	int* kernel_d;
	int* outputMatrix_d;

	// copy the input data from host to GPU device
	copyToDevice<int>(inputMatrix, &inputMatrix_d, inputRow * inputColumn);
	copyToDevice<int>(kernel, &kernel_d, kernelRow * kernelColumn);
	allocateDeviceMemory<int>(&outputMatrix_d, inputRow * inputColumn);

	// allocate memory to hold convoluted matrix in host
	outputMatrix = (int*) malloc( inputRow * inputColumn * sizeof(int));

	start = clock();
	// call the convolution function to be executed on the device
	convoluteAsFilter<<<inputRow, inputColumn>>>(inputMatrix_d, kernel_d, kernelRow, kernelColumn, inputColumn, outputMatrix_d);
	end = clock() - start;

	//copy the output matrix from GOU device to Host
	copyFromDevice<int>(outputMatrix, outputMatrix_d, inputRow * inputColumn);

	//print the output matrix
	printMatrixRowMajor("convoluted matrix from GPU", outputMatrix, inputRow, inputColumn);
	fprintf(fp, "\nGPU time taken = %f\n\n", (double) end / (double) CLOCKS_PER_SEC);
	printf("\nGPU time taken = %f\n\n", (double) end / (double) CLOCKS_PER_SEC);

	//call the convolution function to be executed on host
	start = clock();
	convoluteInHost(inputMatrix, inputRow, inputColumn, kernel, kernelRow, kernelColumn, outputMatrix);
	end = clock() - start;
	//print the output matrix
	printMatrixRowMajor("convoluted matrix from Host", outputMatrix, inputRow, inputColumn);
	printf( "\nPC time taken = %f\n\n", (double) end / (double) CLOCKS_PER_SEC);
	fprintf(fp, "\nPC time taken = %f\n\n", (double) end / (double) CLOCKS_PER_SEC);

	// wait for user key
	getchar(); getchar();

	//free the allocated memory
	free(outputMatrix); free(inputMatrix); free(kernel);
	freeDeviceMemory(3, inputMatrix_d, kernel_d, outputMatrix_d);
	fclose(fp);

	return 0;
}
